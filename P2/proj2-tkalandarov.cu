/* ==================================================================
  Programmer: Timur Kalandarov (tkalandarov@usf.edu)
  An optimized version of SDH algorithm implementation for 3D data
  To run: `/apps/GPU_course/runScript.sh {name.cu} {#of_samples} {bucket_width} {block_size}`
   ==================================================================
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <ctype.h>

#define BOX_SIZE 23000 // size of the data box on one dimension

// definition of single atom
typedef struct atomdesc
{
  double x_pos;
  double y_pos;
  double z_pos;
} atom;

// definition of a bucket
typedef struct hist_entry
{
  unsigned long long distance_count; // need a long long type as the count might be huge
} bucket;

bucket *histogram;                 // list of all buckets in the histogram
unsigned long long PDH_atom_count; // total number of data points
int num_buckets;                   // total number of buckets in the histogram
double PDH_bucket_width;           // value of w
atom *atom_list;                   // list of all data points
int PDH_block_size;                // block size

// These are for an old way of tracking time
struct timezone Idunno;
struct timeval startTime, endTime;

// To track GPU running time
hipEvent_t start, stop;

// set a checkpoint and show the (natural) running time in seconds
void report_running_time1()
{
  long sec_diff, usec_diff;
  gettimeofday(&endTime, &Idunno);
  sec_diff = endTime.tv_sec - startTime.tv_sec;
  usec_diff = endTime.tv_usec - startTime.tv_usec;
  if (usec_diff < 0)
  {
    sec_diff--;
    usec_diff += 1000000;
  }

  long total_usec = sec_diff * 1000000 + usec_diff;
  double total_ms = total_usec / 1000.0; // Convert to milliseconds

  printf("Running time for CPU version: %0.5f ms\n", total_ms);
}

void report_running_time2()
{
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);

  printf("Running time for GPU version: %0.5f ms\n", elapsedTime);
  hipEventDestroy(start);
  hipEventDestroy(stop);
}

// CUDA Error Check
void checkCudaError(hipError_t e, const char *in)
{
  if (e != hipSuccess)
  {
    printf("CUDA Error: %s, %s \n", in, hipGetErrorString(e));
    exit(EXIT_FAILURE);
  }
}

// distance of two points in the atom_list
double p2p_distance1(atom *a, int index1, int index2)
{
  double x1 = a[index1].x_pos;
  double x2 = a[index2].x_pos;

  double y1 = a[index1].y_pos;
  double y2 = a[index2].y_pos;

  double z1 = a[index1].z_pos;
  double z2 = a[index2].z_pos;

  return sqrt((x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) + (z1 - z2) * (z1 - z2));
}

// distance of two points in the atom_list
__device__ double p2p_distance2(atom *a, int index1, int index2)
{
  double x1 = a[index1].x_pos;
  double x2 = a[index2].x_pos;

  double y1 = a[index1].y_pos;
  double y2 = a[index2].y_pos;

  double z1 = a[index1].z_pos;
  double z2 = a[index2].z_pos;

  return sqrt((x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) + (z1 - z2) * (z1 - z2));
}

// brute-force SDH solution in a single CPU thread
int PDH_baseline1()
{
  int i, j, h_pos;
  double dist;

  for (i = 0; i < PDH_atom_count; i++)
  {
    for (j = i + 1; j < PDH_atom_count; j++)
    {
      dist = p2p_distance1(atom_list, i, j);
      h_pos = (int)(dist / PDH_bucket_width);
      histogram[h_pos].distance_count++;
    }
  }
  return 0;
}

__global__ void PDH_baseline2(bucket *histogram, atom *atomList, double bucket_width, unsigned long long PDH_atom_count, int num_buckets)
{
  int tidx = threadIdx.x;                        // Get the thread index within the block
  int threadId = blockIdx.x * blockDim.x + tidx; // Calculate the global thread ID

  extern __shared__ int shared_histogram[]; // Declare shared memory for histogram

  // Initialize shared_histogram to 0, each thread handles a different bucket
  for (int i = tidx; i < num_buckets; i += blockDim.x)
  {
    shared_histogram[i] = 0;
  }

  __syncthreads(); // Ensure all threads have initialized shared_histogram

  int i;
  for (i = threadId + 1; i < PDH_atom_count; i++)
  {
    // Calculate the distance between atomList[i] and atomList[threadId]
    double dist = p2p_distance2(atomList, i, threadId);

    // Determine the histogram bucket (bin) for this distance
    int h_pos = (int)(dist / bucket_width);

    // Atomically increment the corresponding bucket in shared_histogram
    atomicAdd(&(shared_histogram[h_pos]), 1);
  }

  __syncthreads(); // Synchronize threads after histogram computation

  // Accumulate the results from shared_histogram into the global histogram
  for (i = tidx; i < num_buckets; i += blockDim.x)
  {
    // Atomically add the value from shared_histogram to the global histogram
    atomicAdd(&(histogram[i].distance_count), shared_histogram[i]);
  }
}

// print the counts in all buckets of the histogram
void output_histogram(bucket *histogram)
{
  int i;
  unsigned long long total_count = 0;
  for (i = 0; i < num_buckets; i++)
  {
    if (i % 5 == 0) // print 5 buckets in a row
      printf("\n%02d: ", i);
    printf("%15lld ", histogram[i].distance_count);
    total_count += histogram[i].distance_count;

    // we also want to make sure the total distance count is correct
    if (i == num_buckets - 1)
      printf("\n T:%lld \n", total_count);
    else
      printf("| ");
  }
}

void output_histogram_diff(bucket *histogram1, bucket *histogram2)
{
  int i;
  unsigned long long total_count = 0;
  unsigned long long diff;
  for (i = 0; i < num_buckets; i++)
  {
    if (i % 5 == 0) /* we print 5 buckets in a row */
      printf("\n%02d: ", i);
    diff = histogram1[i].distance_count - histogram2[i].distance_count;
    printf("%15lld ", diff);
    total_count += histogram1[i].distance_count;

    // we also want to make sure the total distance count is correct
    if (i == num_buckets - 1)
      printf("\n T:%lld \n", total_count);
    else
      printf("| ");
  }
}

// Input Validation
bool isNumber(char number[], bool isFloat)
{
  for (int i = 0; number[i] != 0; i++)
  {
    if (!isdigit(number[i]))
    {
      if ((number[i] == '.' && isFloat))
      {
        isFloat = false;
      }
      else
      {
        return false;
      }
    }
  }
  return true;
}

int main(int argc, char **argv)
{
  int i;

  if (argc != 4)
  {
    printf("Incorrect number of arguments. Expected: %s <Atom Count> <Bucket Width> <Block Size>\n", argv[0]);
    return EXIT_FAILURE;
  }

  if (isNumber(argv[1], false) && isNumber(argv[2], true) && isNumber(argv[3], false))
  {
    PDH_atom_count = atoi(argv[1]);   // number of atoms
    PDH_bucket_width = atof(argv[2]); // bucket width
    PDH_block_size = atof(argv[3]);   // block size
  }
  else
  {
    printf("Incorrect input. Program will close.\n");
    return EXIT_FAILURE;
  }

  bucket *histogram2;

  num_buckets = (int)(BOX_SIZE * 1.732 / PDH_bucket_width) + 1; // number of buckets needed for SDH

  size_t histogramSize = sizeof(bucket) * num_buckets;
  size_t atomSize = sizeof(atom) * PDH_atom_count;

  histogram = (bucket *)malloc(sizeof(bucket) * num_buckets);
  histogram2 = (bucket *)malloc(sizeof(bucket) * num_buckets);
  atom_list = (atom *)malloc(sizeof(atom) * PDH_atom_count);

  srand(1);
  // generate data following a uniform distribution
  for (i = 0; i < PDH_atom_count; i++)
  {
    atom_list[i].x_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
    atom_list[i].y_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
    atom_list[i].z_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
  }

  // Start CPU timer
  gettimeofday(&startTime, &Idunno);

  PDH_baseline1();

  output_histogram(histogram);
  report_running_time1();

  // Malloc space on device, copy to device
  bucket *d_histogram = NULL;
  atom *d_atom_list = NULL;

  checkCudaError(hipMalloc((void **)&d_histogram, histogramSize),
                 "Malloc Histogram");
  checkCudaError(hipMalloc((void **)&d_atom_list, atomSize),
                 "Malloc Atom List");

  checkCudaError(hipMemcpy(d_histogram, histogram2, histogramSize, hipMemcpyHostToDevice),
                 "Copy histogram to Device");
  checkCudaError(hipMemcpy(d_atom_list, atom_list, atomSize, hipMemcpyHostToDevice),
                 "Copy atom_list to Device");

  // Start GPU timer
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // CUDA Kernel Call
  int numBlocks = (PDH_atom_count + PDH_block_size - 1) / PDH_block_size;
  int sharedMemorySize = sizeof(int) * num_buckets;
  PDH_baseline2<<<numBlocks, PDH_block_size, sharedMemorySize>>>(d_histogram, d_atom_list, PDH_bucket_width, PDH_atom_count, num_buckets);

  checkCudaError(hipGetLastError(), "Kernel Launch");

  checkCudaError(hipMemcpy(histogram2, d_histogram, histogramSize, hipMemcpyDeviceToHost),
                 "Copy device histogram to host");

  output_histogram(histogram2);
  report_running_time2();

  // Show differences in two histograms
  output_histogram_diff(histogram, histogram2);

  checkCudaError(hipFree(d_histogram), "Free device histogram");
  checkCudaError(hipFree(d_atom_list), "Free device atom_list");

  checkCudaError(hipDeviceReset(), "Device reset");

  free(histogram);
  free(histogram2);
  free(atom_list);

  return 0;
}