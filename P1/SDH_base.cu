/* ==================================================================
  Programmer: Timur Kalandarov (tkalandarov@usf.edu)
  The basic SDH algorithm implementation for 3D data
  To run: `/apps/GPU_course/runScript.sh /home/t/tkalandarov/COP4520/P1/SDH.cu 10000 500`
   ==================================================================
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BOX_SIZE 23000 // size of the data box on one dimension

// definition of single atom
typedef struct atomdesc
{
  double x_pos;
  double y_pos;
  double z_pos;
} atom;

// definition of a bucket
typedef struct hist_entry
{
  unsigned long long distance_count; // need a long long type as the count might be huge
} bucket;

bucket *histogram;                 // list of all buckets in the histogram
unsigned long long PDH_atom_count; // total number of data points
int num_buckets;                   // total number of buckets in the histogram
double PDH_bucket_width;           // value of w
atom *atom_list;                   // list of all data points

// CUDA Error Check
void checkCudaError(hipError_t e, const char *in)
{
  if (e != hipSuccess)
  {
    printf("CUDA Error: %s, %s \n", in, hipGetErrorString(e));
    exit(EXIT_FAILURE);
  }
}

// distance of two points in the atom_list
__device__ double p2p_distance(atom *a, int index1, int index2)
{
  double x1 = a[index1].x_pos;
  double x2 = a[index2].x_pos;

  double y1 = a[index1].y_pos;
  double y2 = a[index2].y_pos;

  double z1 = a[index1].z_pos;
  double z2 = a[index2].z_pos;

  return sqrt((x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) + (z1 - z2) * (z1 - z2));
}

__global__ void PDH_baseline(bucket *histogram, atom *atomList, double bucket_width, unsigned long long PDH_atom_count)
{
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;

  if (threadId >= PDH_atom_count)
    return;

  for (int i = threadId + 1; i < PDH_atom_count; i++)
  {
    double dist = p2p_distance(atomList, threadId, i);
    int pos = (int)(dist / bucket_width);

    atomicAdd(&histogram[pos].distance_count, 1);
  }
}

// print the counts in all buckets of the histogram
void output_histogram(bucket *histogram)
{
  int i;
  unsigned long long total_count = 0;
  for (i = 0; i < num_buckets; i++)
  {
    if (i % 5 == 0) // print 5 buckets in a row
      printf("\n%02d: ", i);
    printf("%15lld ", histogram[i].distance_count);
    total_count += histogram[i].distance_count;

    // we also want to make sure the total distance count is correct
    if (i == num_buckets - 1)
      printf("\n T:%lld \n", total_count);
    else
      printf("| ");
  }
}

int main(int argc, char **argv)
{
  int i;

  PDH_atom_count = atoi(argv[1]);   // number of atoms
  PDH_bucket_width = atof(argv[2]); // input distance: bucket width

  num_buckets = (int)(BOX_SIZE * 1.732 / PDH_bucket_width) + 1; // number of buckets needed for SDH

  size_t histogramSize = sizeof(bucket) * num_buckets;
  size_t atomSize = sizeof(atom) * PDH_atom_count;

  histogram = (bucket *)malloc(sizeof(bucket) * num_buckets);
  atom_list = (atom *)malloc(sizeof(atom) * PDH_atom_count);

  srand(1);
  // generate data following a uniform distribution
  for (i = 0; i < PDH_atom_count; i++)
  {
    atom_list[i].x_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
    atom_list[i].y_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
    atom_list[i].z_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
  }

  // Malloc space on device, copy to device
  bucket *d_histogram = NULL;
  atom *d_atom_list = NULL;

  checkCudaError(hipMalloc((void **)&d_histogram, histogramSize),
                 "Malloc Histogram");
  checkCudaError(hipMalloc((void **)&d_atom_list, atomSize),
                 "Malloc Atom List");

  checkCudaError(hipMemcpy(d_histogram, histogram, histogramSize, hipMemcpyHostToDevice),
                 "Copy histogram to Device");
  checkCudaError(hipMemcpy(d_atom_list, atom_list, atomSize, hipMemcpyHostToDevice),
                 "Copy atom_list to Device");

  // CUDA Kernel Call
  PDH_baseline<<<ceil(PDH_atom_count / 32), 32>>>(d_histogram, d_atom_list, PDH_bucket_width, PDH_atom_count);
  checkCudaError(hipGetLastError(), "Kernel Launch");

  checkCudaError(hipMemcpy(histogram, d_histogram, histogramSize, hipMemcpyDeviceToHost),
                 "Copy device histogram to host");

  output_histogram(histogram);

  checkCudaError(hipFree(d_histogram), "Free device histogram");
  checkCudaError(hipFree(d_atom_list), "Free device atom_list");

  free(histogram);
  free(atom_list);

  checkCudaError(hipDeviceReset(), "Device reset");

  return 0;
}
